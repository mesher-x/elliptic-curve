#include "kernels.cuh"

#include <cassert>
#include <cmath>
#include <cstring>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <sstream>
#include <tuple>
#include <vector>

const uchar scalar_size = 8;
const uchar b2_len = 65;

void fst(int argc, char *argv[]);
//void snd(int argc, char *argv[]);

int main(int argc, char *argv[])
{
    fst(argc, argv);
    //snd(argc, argv);
    return 0;
}

void convert_input_hex_to_scalar(const char *hex, u32 *array);

void fst(int argc, char *argv[])
{
    if (argc < 2) {
        std::cout << "Please provide a hex string as a command line argument." << std::endl;
        return;
    }

    if (argc > 2) {
        std::cout << "command line argument must be single" << std::endl;
        return;
    }

    if (strlen(argv[1]) > 64) {
        std::cout << "hex string must be 64 chars or shorter" << std::endl;
        return;
    }

    u32 scalar[scalar_size];
    convert_input_hex_to_scalar(argv[1], scalar);

    u32* scalar_d;
    hipError_t e;
    e = hipMalloc((void**)&scalar_d, scalar_size * sizeof(u32));
    assert(e == hipSuccess);
    e = hipMemcpy(scalar_d, scalar, scalar_size * sizeof(u32), hipMemcpyHostToDevice);
    assert(e == hipSuccess);

    uchar* b2_d;
    e = hipMalloc((void**)&b2_d, b2_len * sizeof(uchar));
    assert(e == hipSuccess);
    
    uint64_t* r_d;
    e = hipMalloc((void**)&r_d, 5 * sizeof(uint64_t));
    assert(e == hipSuccess);

    uint64_t* a_d;
    e = hipMalloc((void**)&a_d, 5 * sizeof(uint64_t));
    assert(e == hipSuccess);

    fst_kernel(b2_d, scalar_d, r_d, a_d);

    uchar b2[b2_len];
    e = hipMemcpy(b2, b2_d, b2_len * sizeof(uchar), hipMemcpyDeviceToHost);
    assert(e == hipSuccess);

    uint64_t r[5];
    e = hipMemcpy(r, r_d, 5 * sizeof(uint64_t), hipMemcpyDeviceToHost);
    assert(e == hipSuccess);

    for (int i = 0; i < 5; i++)
    {
        std::cout << ((r[i] >> 32) & 0xFFFFFFFF) << (r[i] & 0xFFFFFFFF);
    }
    std::cout << std::endl;

    e = hipFree(scalar_d);
    assert(e == hipSuccess);
    e = hipFree(b2_d);
    assert(e == hipSuccess);
    e = hipFree(r_d);
    assert(e == hipSuccess);
    e = hipFree(a_d);
    assert(e == hipSuccess);
    
    std::stringstream ss;
    ss << "0x";
    ss << std::hex << std::setfill('0');
    for (int i = 0; i < b2_len; i++)
    {
        ss << std::setw(2) << (u32)b2[i];
    }
    std::cout << "result:" << std::endl << ss.str() << std::endl;
}

void convert_input_hex_to_scalar(const char *hex, u32 *array) {
    std::string hex_str(hex);
    assert(hex_str.length() <= 64);
    std::string paddedHex = std::string(64 - hex_str.length(), '0') + hex_str;

    for (int i = 0; i < 8; i++)
    {
        std::string sub = paddedHex.substr(i * 8, 8);
        array[7 - i] = std::stoul(sub, nullptr, 16);
    }
}

// int main()
// {
//     int a = 3, b = 4, c;
//     int* d_c;

//     // Allocate memory on the device
//     hipMalloc((void**)&d_c, sizeof(int));

//     // Launch the kernel on the device
//     add_kernel(a, b, d_c);

//     // Copy the result back to the host
//     hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

//     // Free the device memory
//     hipFree(d_c);

//     // Print the result
//     std::cout << "The sum of " << a << " and " << b << " is " << c << std::endl;

//     return 0;
// }
